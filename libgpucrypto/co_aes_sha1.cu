#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <assert.h>

#include "aes_core.h"
#include "sha1.h"
#include "crypto_size.h"

/* AES counter mode + HMAC SHA-1, 
   the encryption of each block in AES counter mode is not parallelized in this implementation */
__global__ void aes_ctr_sha1_kernel(
			uint8_t	*input_buf,
			uint8_t *output_buf,
			const uint8_t *aes_keys,
			uint8_t *ivs,
			const uint8_t *hmac_keys,
			const uint32_t *pkt_offset,
			const uint16_t *length,
			const unsigned int num_flows,
			uint8_t *checkbits)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint16_t len;

/**************************************************************************
  AES Encryption is started first
 ***************************************************************************/
#if 1
	__shared__ uint32_t shared_Te0[256];
	__shared__ uint32_t shared_Te1[256];
	__shared__ uint32_t shared_Te2[256];
	__shared__ uint32_t shared_Te3[256];
	__shared__ uint32_t shared_Rcon[10];

	/* Private counter 128 bits */
	uint32_t keystream[4];

	/* initialize T boxes */
	for (unsigned i = 0; i * blockDim.x < 256 ; i ++) {
		unsigned index = threadIdx.x + i * blockDim.x;
		if (index >= 256)
			break;
		shared_Te0[index] = Te0_ConstMem[index];
		shared_Te1[index] = Te1_ConstMem[index];
		shared_Te2[index] = Te2_ConstMem[index];
		shared_Te3[index] = Te3_ConstMem[index];
	}

	for(unsigned i = 0; i * blockDim.x < 10; i ++){
		int index = threadIdx.x + blockDim.x * i;
		if(index < 10){
			shared_Rcon[index] = rcon[index];
		}
	}

	/* ----debug-----*/
	if (idx >= num_flows) {
		//printf("idx = %d, num_flows = %d, exit.\n", idx, num_flows);
		return;
	}

	/* make sure T boxes have been initialized. */
	__syncthreads();

	/* Encrypt using counter mode, this is the actual length of the packet */
	/* pkt_offset[idx + 1] - pkt_offset[idx] is used for "length[idx] + padding for HMAC + HMAC sha-1 tag" */
	len = length[idx];

	/* Skip RTP header to Locate the data to be encrypted */
	uint8_t *in = pkt_offset[idx] + input_buf;
	uint8_t cc = in[0] & 0x0F; /* Get the number of CSRC identifiers */
	if (cc != 0) {
		//printf("%d,%d,%d,%d\n", in[0], in[1], in[2], in[3]);
		return;
	}
	uint16_t header_len = 12 + 4 * cc; /* Get the total header length */

	/* Jump to the parts need encryption */
	in = in + header_len; /* Get to the payload */

	uint8_t *out = pkt_offset[idx] + output_buf;
	out	= out + header_len; /* Get to the payload */
	len	-= header_len; /* data length that needs encryption */
	
	assert(out == in);

	/* ----debug----- */
	if (len <= 0) {
		printf("idx = %d, len <= 0, exit.\n", idx);
		return;
	}

	const uint8_t *key = idx * 16 + aes_keys;
	uint64_t *iv = (uint64_t *) (idx * AES_BLOCK_SIZE + ivs);
	// printf("idx %d is writing : %d header_len : %d len\n", idx, header_len, len);
	while (len >= AES_BLOCK_SIZE) {

		/* for the ith block, its input is ((iv + i) mod 2^128)*/
		iv[0] ++;
		if (iv[0] == 0)
			iv[1] ++;

		/* Get the keystream here */
		AES_128_encrypt((uint8_t *)iv, (uint8_t *)keystream, key,
				shared_Te0, shared_Te1, shared_Te2, shared_Te3, shared_Rcon);

		*((uint32_t*)out)       = *((uint32_t*)in)       ^ *((uint32_t*)keystream);
		*(((uint32_t*)out) + 1) = *(((uint32_t*)in) + 1) ^ *(((uint32_t*)keystream) + 1);
		*(((uint32_t*)out) + 2) = *(((uint32_t*)in) + 2) ^ *(((uint32_t*)keystream) + 2);
		*(((uint32_t*)out) + 3) = *(((uint32_t*)in) + 3) ^ *(((uint32_t*)keystream) + 3);

		//if (idx == 0)
		//	printf("in = %p, out = %p, output_buf = %p, offset = %d\n", in, out, output_buf, out - output_buf);

		len -= AES_BLOCK_SIZE;
		in  += AES_BLOCK_SIZE;
		out += AES_BLOCK_SIZE;

		//if (idx == 0)
		//	printf("len = %d,  %d\n", len, AES_BLOCK_SIZE);
	}

	if (len) {
		//if (idx == 0)
		//	printf("len = %d\n");
		/* for the ith block, its input is ((iv + i) mod 2^128)*/
		iv[0] ++;
		if (iv[0] == 0)
			iv[1] ++;

		AES_128_encrypt((uint8_t *)iv, (uint8_t *)keystream, key,
				shared_Te0, shared_Te1, shared_Te2, shared_Te3, shared_Rcon);

		for(unsigned n = 0; n < len; ++n)
			out[n] = in[n] ^ ((uint8_t *)keystream)[n];
	}

	__syncthreads();
#endif
/**************************************************************************
  AES Encryption completed, Now we start SHA-1 Calculation
 ***************************************************************************/
#if 0
	uint32_t w_register[16];

	uint32_t *w = w_register;
	hash_digest_t h;
	uint32_t offset = pkt_offset[idx];
	len = length[idx];

	//uint16_t sha1_pad_len = (len + 8 + 1 + 63) & (~0x3F);
	//sha1_pad_len -= 64;
	uint16_t sha1_output_pos = (len + 3) & ~0x03;
	uint32_t *sha1_out = (uint32_t *)(input_buf + offset + sha1_output_pos);

	for (unsigned i = 0; i < 16; i++)
		w[i] = 0x36363636;

	/* In SRTP, HMAC_KEY_SIZE is 160 bits = 20 bytes */
	xorpads(w, (uint32_t *)(hmac_keys + HMAC_KEY_SIZE * idx));

	h.h1 = 0x67452301;
	h.h2 = 0xEFCDAB89;
	h.h3 = 0x98BADCFE;
	h.h4 = 0x10325476;
	h.h5 = 0xC3D2E1F0;

	//SHA1 compute on ipad
	computeSHA1Block((char*)w, w, 0, 64, h);

	//SHA1 compute on message
	unsigned num_iter = (len + 8 + 1 + 63) >> 6; // the same as divide 64
	for (unsigned i = 0; i < num_iter; i ++)
		computeSHA1Block((char *)(input_buf + offset), w, i * 64, len, h);

	/* In SRTP, sha1_out has only 80 bits output 32+32+16 = 80 */
	*(sha1_out)   = swap(h.h1);
	*(sha1_out+1) = swap(h.h2);
	uint32_t temp = swap(h.h3);
	*(uint16_t *)(sha1_out+2)  = ((uint16_t *)&temp)[0];

	h.h1 = 0x67452301;
	h.h2 = 0xEFCDAB89;
	h.h3 = 0x98BADCFE;
	h.h4 = 0x10325476;
	h.h5 = 0xC3D2E1F0;

	for (unsigned i = 0; i < 16; i++)
		w[i] = 0x5c5c5c5c;

	xorpads(w, (uint32_t*)(hmac_keys + 20 * idx));

	//SHA 1 compute on opads
	computeSHA1Block((char*)w, w, 0, 64, h);

	//SHA 1 compute on (hash of ipad|m)
	//HMAC_TAG_SIZE  = 10
	computeSHA1Block((char*)sha1_out, w, 0, 10, h);

	*(sha1_out)   = swap(h.h1);
	*(sha1_out+1) = swap(h.h2);
	temp = swap(h.h3);
	*(uint16_t *)(sha1_out+2)  = ((uint16_t *)&temp)[0];

	__syncthreads();
#endif
	// Now we set the checkbits
	//*(checkbits + idx) = 1;
	return;
}

extern "C" void co_aes_sha1_gpu(
			uint8_t		*in,
			uint8_t		*out,
			uint8_t		*aes_keys,
			uint8_t		*ivs,
			uint8_t		*hmac_keys,
			uint32_t	*pkt_offset,
			uint16_t	*actual_length,
			unsigned int num_flows,
			uint8_t		*checkbits,
			unsigned	threads_per_blk,
			hipStream_t stream)
{
	int num_blks = (num_flows + threads_per_blk - 1) / threads_per_blk;

	//printf("stream=%d, threads_per_blk =%d, num_blks = %d\n", stream, threads_per_blk, num_blks);
	if (stream == 0) {
		aes_ctr_sha1_kernel<<<num_blks, threads_per_blk>>>(
		       in, out, aes_keys, ivs, hmac_keys, pkt_offset, actual_length, num_flows, checkbits);
	} else  {
		aes_ctr_sha1_kernel<<<num_blks, threads_per_blk, 0, stream>>>(
		       in, out, aes_keys, ivs, hmac_keys, pkt_offset, actual_length, num_flows, checkbits);
	}
}

